/*-----------------------------------------------
 * 请在此处填写你的个人信息
 * 学号: SA24218140
 * 姓名: 鲍习坤
 * 邮箱: 2363810524@qq.com
 ------------------------------------------------*/


#include <hip/hip_runtime.h>
#include <chrono>
#include <cstring>
#include <fstream>
#include <iostream>
#include <string>

#define AT(x, y, z) universe[(x) * N * N + (y) * N + z]

using namespace std;

// 存活细胞数
int population(int N, char *universe)
{
    int result = 0;
    for (int i = 0; i < N * N * N; i++)
        result += universe[i];
    return result;
}

// 打印世界状态
void print_universe(int N, char *universe)
{
    // 仅在N较小(<= 32)时用于Debug
    if (N > 32)
        return;
    for (int x = 0; x < N; x++)
    {
        for (int y = 0; y < N; y++)
        {
            for (int z = 0; z < N; z++)
            {
                if (AT(x, y, z))
                    cout << "O ";
                else
                    cout << "* ";
            }
            cout << endl;
        }
        cout << endl;
    }
    cout << "population: " << population(N, universe) << endl;
}

// 核心计算代码，将世界向前推进T个时刻
void life3d_run_cpu(int N, char *universe, int T)
{
    char *next = (char *)malloc(N * N * N);
    for (int t = 0; t < T; t++)
    {
        // outerloop: iter universe
        for (int x = 0; x < N; x++)
            for (int y = 0; y < N; y++)
                for (int z = 0; z < N; z++)
                {
                    // inner loop: stencil
                    int alive = 0;
                    for (int dx = -1; dx <= 1; dx++)
                        for (int dy = -1; dy <= 1; dy++)
                            for (int dz = -1; dz <= 1; dz++)
                            {
                                if (dx == 0 && dy == 0 && dz == 0)
                                    continue;
                                int nx = (x + dx + N) % N;
                                int ny = (y + dy + N) % N;
                                int nz = (z + dz + N) % N;
                                alive += AT(nx, ny, nz);
                            }
                    if (AT(x, y, z) && (alive < 5 || alive > 7))
                        next[x * N * N + y * N + z] = 0;
                    else if (!AT(x, y, z) && alive == 6)
                        next[x * N * N + y * N + z] = 1;
                    else
                        next[x * N * N + y * N + z] = AT(x, y, z);
                }
        memcpy(universe, next, N * N * N);
    }
    free(next);
}

// CUDA核函数
__global__ void life3d_kernel(int N, char *universe, char *next) {
    // 计算线程负责得点坐标
    int x = blockIdx.x + blockDim.x + threadIdx.x;
    int y = blockIdx.y + blockDim.y + threadIdx.y;
    int z = blockIdx.z + blockDim.z + threadIdx.z;

    if (x >= N || y >= N || z >= N) return;

    int alive = 0;

    for (int dx = -1; dx <= 1; dx++) {
        for (int dy = -1; dy <= 1; dy++) {
            for (int dz = -1; dz <= 1; dz++) {
                if (dx == 0 && dy == 0 && dz ==0) continue;

                int nx = (x + dx + N) % N;
                int ny = (y + dy + N) % N;
                int nz = (z + dz + N) % N;

                alive += universe[nx * N * N + ny * y + nz];
            }
        }
    }


    int index = x * N * N + y * N + z;
    if (universe[index] && (alive < 5 || alive > 7)) next[index] = 0;
    else if (!universe[index] && alive == 6) next[index] = 1;
    else next[index] = universe[index];

}

void life3d_run_gpu(int N, char *universe, int T) {
    char *d_universe, *d_next;

    // 分配GPU内存
    hipMalloc((void **)&d_universe, N * N * N * sizeof(char));
    hipMalloc((void **)&d_next, N * N * N * sizeof(char));

    // 在GPU上初始化数据
    hipMemcpy(d_universe, universe, N * N * N * sizeof(char), hipMemcpyHostToDevice);

    // 定义线程块和网络维度
    int threadPerBlock = 8;
    dim3 blockDim(threadPerBlock, threadPerBlock, threadPerBlock);
    dim3 gridDim((N + blockDim.x - 1) / blockDim.x,
                 (N + blockDim.y - 1) / blockDim.y,
                 (N + blockDim.z - 1) / blockDim.z);

    for (int t = 0; t < T; t++) {
        life3d_kernel<<<gridDim, blockDim>>>(N, d_universe, d_next);

        char *temp = d_universe;
        d_universe = d_next;
        d_next = temp;
    }
    hipMemcpy(universe, d_universe, N * N * N * sizeof(char), hipMemcpyDeviceToHost);

    hipFree(d_universe);
    hipFree(d_next);
}

// 读取输入文件
void read_file(char *input_file, char *buffer)
{
    ifstream file(input_file, std::ios::binary | std::ios::ate);
    if (!file.is_open())
    {
        cout << "Error: Could not open file " << input_file << std::endl;
        exit(1);
    }
    std::streamsize file_size = file.tellg();
    file.seekg(0, std::ios::beg);
    if (!file.read(buffer, file_size))
    {
        std::cerr << "Error: Could not read file " << input_file << std::endl;
        exit(1);
    }
    file.close();
}

// 写入输出文件
void write_file(char *output_file, char *buffer, int N)
{
    ofstream file(output_file, std::ios::binary | std::ios::trunc);
    if (!file)
    {
        cout << "Error: Could not open file " << output_file << std::endl;
        exit(1);
    }
    file.write(buffer, N * N * N);
    file.close();
}

int main(int argc, char **argv)
{
    // cmd args
    if (argc < 5)
    {
        cout << "usage: ./life3d N T input output" << endl;
        return 1;
    }
    int N = std::stoi(argv[1]);
    int T = std::stoi(argv[2]);
    char *input_file = argv[3];
    char *output_file = argv[4];

    char *universe = (char *)malloc(N * N * N);
    read_file(input_file, universe);

    int start_pop = population(N, universe);
    auto start_time = std::chrono::high_resolution_clock::now();
    life3d_run_gpu(N, universe, T);
    auto end_time = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> duration = end_time - start_time;
    int final_pop = population(N, universe);
    write_file(output_file, universe, N);

    cout << "start population: " << start_pop << endl;
    cout << "final population: " << final_pop << endl;
    double time = duration.count();
    cout << "time: " << time << "s" << endl;
    cout << "cell per sec: " << T / time * N * N * N << endl;

    free(universe);
    return 0;
}